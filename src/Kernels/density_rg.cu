#include "hip/hip_runtime.h"
#include "../SDDK/GPU/cuda_common.hpp"

__global__ void update_density_rg_1_gpu_kernel(int size__,
                                               hipDoubleComplex const* psi_rg__,
                                               double wt__,
                                               double* density_rg__)
{
    int ir = blockIdx.x * blockDim.x + threadIdx.x;
    if (ir < size__)
    {
        hipDoubleComplex z = psi_rg__[ir];
        density_rg__[ir] += (z.x * z.x + z.y * z.y) * wt__;
    }
}

extern "C" void update_density_rg_1_gpu(int size__, 
                                        hipDoubleComplex const* psi_rg__, 
                                        double wt__, 
                                        double* density_rg__)
{
    //CUDA_timer t("update_density_rg_1_gpu");

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(size__, grid_t.x));

    update_density_rg_1_gpu_kernel <<<grid_b, grid_t>>>
    (
        size__,
        psi_rg__,
        wt__,
        density_rg__
    );
}

__global__ void update_density_rg_2_gpu_kernel(int size__,
                                               hipDoubleComplex const* psi_up_rg__,
                                               hipDoubleComplex const* psi_dn_rg__,
                                               double wt__,
                                               double* density_x_rg__,
                                               double* density_y_rg__)
{
    int ir = blockIdx.x * blockDim.x + threadIdx.x;
    if (ir < size__) {
        hipDoubleComplex z = hipCmul(psi_up_rg__[ir], hipConj(psi_dn_rg__[ir]));
        density_x_rg__[ir] += 2 * z.x * wt__;
        density_y_rg__[ir] -= 2 * z.y * wt__;
    }
}

extern "C" void update_density_rg_2_gpu(int size__, 
                                        hipDoubleComplex const* psi_up_rg__, 
                                        hipDoubleComplex const* psi_dn_rg__, 
                                        double wt__, 
                                        double* density_x_rg__,
                                        double* density_y_rg__)
{
    //CUDA_timer t("update_density_rg_1_gpu");

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(size__, grid_t.x));

    update_density_rg_2_gpu_kernel <<<grid_b, grid_t>>>
    (
        size__,
        psi_up_rg__,
        psi_dn_rg__,
        wt__,
        density_x_rg__,
        density_y_rg__
    );
}


