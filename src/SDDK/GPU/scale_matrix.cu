#include "hip/hip_runtime.h"
#include "cuda_common.h"

__global__ void scale_matrix_columns_gpu_kernel
(
    int nrow,
    hipDoubleComplex* mtrx,
    double* a
)
{
    int icol = blockIdx.y;
    int irow = blockIdx.x * blockDim.x + threadIdx.x;
    if (irow < nrow) 
    {
        mtrx[array2D_offset(irow, icol, nrow)] =
            hipCmul(mtrx[array2D_offset(irow, icol, nrow)], make_hipDoubleComplex(a[icol], 0));
    }
}

// scale each column of the matrix by a column-dependent constant
extern "C" void scale_matrix_columns_gpu(int nrow,
                                         int ncol,
                                         hipDoubleComplex* mtrx,
                                         double* a)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nrow, grid_t.x), ncol);

    scale_matrix_columns_gpu_kernel <<<grid_b, grid_t>>>
    (
        nrow,
        mtrx,
        a
    );
}

__global__ void scale_matrix_rows_gpu_kernel
(
    int nrow__,
    hipDoubleComplex* mtrx__,
    double const* v__
)
{
    int icol = blockIdx.y;
    int irow = blockDim.x * blockIdx.x + threadIdx.x;
    if (irow < nrow__) {
        hipDoubleComplex z = mtrx__[array2D_offset(irow, icol, nrow__)];
        mtrx__[array2D_offset(irow, icol, nrow__)] = make_hipDoubleComplex(z.x * v__[irow], z.y * v__[irow]);
    }
}

// scale each row of the matrix by a row-dependent constant
extern "C" void scale_matrix_rows_gpu(int nrow__,
                                      int ncol__,
                                      hipDoubleComplex* mtrx__,
                                      double const* v__)
{
    dim3 grid_t(256);
    dim3 grid_b(num_blocks(nrow__, grid_t.x), ncol__);

    scale_matrix_rows_gpu_kernel <<<grid_b, grid_t>>>
    (
        nrow__,
        mtrx__,
        v__
    );
}

__global__ void scale_matrix_elements_gpu_kernel
(
    hipDoubleComplex* mtrx__,
    int ld__,
    int nrow__,
    double beta__
)
{
    int icol = blockIdx.y;
    int irow = blockDim.x * blockIdx.x + threadIdx.x;
    if (irow < nrow__) {
        hipDoubleComplex z = mtrx__[array2D_offset(irow, icol, ld__)];
        mtrx__[array2D_offset(irow, icol, ld__)] = make_hipDoubleComplex(z.x * beta__, z.y * beta__);
    }
}

extern "C" void scale_matrix_elements_gpu(hipDoubleComplex* ptr__,
                                          int ld__,
                                          int nrow__,
                                          int ncol__,
                                          double beta__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nrow__, grid_t.x), ncol__);

    scale_matrix_elements_gpu_kernel <<<grid_b, grid_t>>>
    (
        ptr__,
        ld__,
        nrow__,
        beta__
    );
}
